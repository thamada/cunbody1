#include "hip/hip_runtime.h"
// Time-stamp: <2009-01-17 15:21:58 hamada>

/*
 * Copyright (C) 2007 
 *      Tsuyoshi Hamada <hamada@progrape.jp>
 *      All rights reserved.
 * This code is released under version 2 of the GNU GPL.
 */

#define IDIM  (4)
#define JDIM  (4)
#define FDIM  (3)
#include "vforce.h"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <cutil.h>

//#undef SM_MAX_BYTE
//#define SM_MAX_BYTE (16384-32)
//#define NJ_SHMEM 256
#define NJ_SHMEM 128  // ** Technic ** 
#define NSP 8
#define NVSP 16
//#define NVSP 24 // GTX260
#define NPIPE (NSP*NVSP)
#define   KIRIAGE(x,y)     (((x) % (y)) ?  ((x/y)+1) : (x/y))
#define   MAX(x,y)     (((x) > (y)) ?  (x) : (y))
#define   MIN(x,y)     (((x) < (y)) ?  (x) : (y))

//#include "cunbody_dbg.h"


//#include "cunbody_kernel.cc"
#include "cunbody_kernel_00.cc"
#include "cunbody_kernel_01.cc"
#include "cunbody_kernel_basic.cc"


namespace libcunbody{
  using namespace std;

  class cunbody1_bench
  {
  private:
    bool is_open;
    int devid;
    char gpu_name[128];
    float4* h_xj;
    float* h_xi;
    float* h_fo;
    float4* d_xj;
    float* d_xi;
    float* d_fo;

    void dev_check(void){
      int ndev;
      CUDA_SAFE_CALL(hipSetDevice(devid));
      CUDA_SAFE_CALL(hipGetDeviceCount(&ndev));
      if(ndev == 0){
	fprintf(stdout, "ndev = %d @ %s|%d\n", ndev, __FILE__, __LINE__);
	exit(-1);
	/*
	  }else if(ndev > 1){
	  fprintf(stdout, "ndev = %d @ %s|%d\n", ndev, __FILE__, __LINE__);
	  fprintf(stdout, "This library doesn't work with multiple GPUs.\n");
	  exit(-1);
	*/
      }else{
	int dev = devid;
	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev));
	if (deviceProp.major == 9999 && deviceProp.minor == 9999){
	  printf("There is no device supporting CUDA.\n");
	}
	sprintf(gpu_name, "%s", deviceProp.name);
	printf("  GPU : %s\n",                       deviceProp.name);
	printf("  Major # :  %d\n",                  deviceProp.major);
	printf("  Minor # :  %d\n",                  deviceProp.minor);
	printf("  core clock rate :  %.2f GHz\n",    deviceProp.clockRate * 1e-6f);
#if  (CUDART_VERSION >= 2000)
	printf("  Number of cores :  %d\n",      8 * deviceProp.multiProcessorCount);
	printf("  Number of multiprocessors : %d\n", deviceProp.multiProcessorCount);
#endif
	printf("  global memory : %u bytes\n",    deviceProp.totalGlobalMem);
	printf("  constant memory : %u bytes\n",         deviceProp.totalConstMem); 
	printf("  shared memory per block : %u bytes\n", deviceProp.sharedMemPerBlock);
	printf("  registers available per block : %d\n", deviceProp.regsPerBlock);
	printf("  Warp size : %d\n", deviceProp.warpSize);
	printf("  Max # of threads per block : %d\n",   deviceProp.maxThreadsPerBlock);
	printf("  Max sizes of each dimension of a block : %d x %d x %d\n",
	       deviceProp.maxThreadsDim[0],
	       deviceProp.maxThreadsDim[1],
	       deviceProp.maxThreadsDim[2]);
	printf("  Maximum sizes of each dimension of a grid : %d x %d x %d\n",
	       deviceProp.maxGridSize[0],
	       deviceProp.maxGridSize[1],
	       deviceProp.maxGridSize[2]);
	printf("  Maximum memory pitch : %u bytes\n", deviceProp.memPitch);
	printf("  Texture alignment    : %u bytes\n", deviceProp.textureAlignment);
#if  (CUDART_VERSION >= 2000)
	printf("  Concurrent copy and execution: %s\n", deviceProp.deviceOverlap ? "Yes" : "No");
#endif
	printf("  hipSetDevice to %d \n", dev);
      }
    }

  public:

    cunbody1_bench() {
      is_open = false;
      devid = 0;
    }

    ~cunbody1_bench() {
      this->close();
      devid = 0;
      is_open = false;
    }

    /* no need to call if you wan't*/
    void close(void)
    {
      // cleanup memory
      CUDA_SAFE_CALL(hipHostFree(h_xj));
      CUDA_SAFE_CALL(hipHostFree(h_xi));
      CUDA_SAFE_CALL(hipHostFree(h_fo));
      CUDA_SAFE_CALL(hipFree(d_xi));
      CUDA_SAFE_CALL(hipFree(d_fo));
      CUDA_SAFE_CALL(hipFree(d_xj));
    }

    void set_devid(int id)
    {
      devid = id;
    }

    void force(double xj[][3], double mj[], double xi[][3], double eps2, double a[][3], int ni, int nj)
    {
      int nj1 = ((nj+NJ_SHMEM-1)/NJ_SHMEM)*NJ_SHMEM;

      if( ni > (0x1<<17) ){
	printf("ERROR %s|%d\n",__FILE__, __LINE__);
	printf(" ni > 131072 : ni=%d\n",ni);
	exit(-1);
      }

      if( nj > (0x1<<17) ){
	printf("ERROR %s|%d\n",__FILE__, __LINE__);
	printf(" nj > 131072 : nj=%d\n",nj);
	exit(-1);
      }

      unsigned int ip_size = sizeof(float) * ni * IDIM;
      unsigned int jp_size = sizeof(float4) * nj1;
      unsigned int fo_size = sizeof(float) * ni * FDIM;

      if(is_open == false){
	//    CUT_DEVICE_INIT();
	//    CUT_CHECK_DEVICE();
	unsigned int _nmax = 1<<17; // 131072
	//    unsigned int _nmax = 1<<14; // 16384
	unsigned int _ip_size = sizeof( float) * _nmax * IDIM;
	unsigned int _jp_size = sizeof(float4) * _nmax;
	unsigned int _fo_size = sizeof(float) * _nmax * FDIM;

	dev_check();

	CUDA_SAFE_CALL(  hipHostMalloc( (void**)&h_xj, _jp_size)  );
	CUDA_SAFE_CALL(  hipHostMalloc( (void**)&h_xi, _ip_size)  );
	CUDA_SAFE_CALL(  hipHostMalloc( (void**)&h_fo, _fo_size)  );

	CUDA_SAFE_CALL( hipMalloc( (void**) &d_xj, _jp_size));
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_xi, _ip_size));
	CUDA_SAFE_CALL( hipMalloc( (void**) &d_fo, _fo_size));

	for(int i = 0; i < _nmax ; i++) h_xj[i] = make_float4(0.0, 0.0, 0.0, 0.0);
	CUDA_SAFE_CALL( hipMemcpy( d_xj, h_xj, _jp_size, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy( d_xi, h_xj, _ip_size, hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy( d_fo, h_xj, _fo_size, hipMemcpyHostToDevice) );
	fprintf(stderr, "open %s by CUNBODY-1 library: rev.hamada20080905  (^<_^)/ %d\n", gpu_name, devid);
	is_open = true;
      }
      // ------------------------------------------------------------ xj
      for(int i = 0; i < nj; i++){
	h_xj[i].x = (float) xj[i][0];
	h_xj[i].y = (float) xj[i][1];
	h_xj[i].z = (float) xj[i][2];
	h_xj[i].w = (float) mj[i];
      }

      if(nj < nj1){
	for(int i = nj; i < nj1; i++)  h_xj[i] = make_float4(0.0, 0.0, 0.0, 0.0);
      }

      CUDA_SAFE_CALL( hipMemcpy( d_xj, h_xj, jp_size, hipMemcpyHostToDevice) );

      // ------------------------------------------------------------ xi
      for(int i = 0; i < ni; i++){
	h_xi[i     ] = (float) xi[i][0];
	h_xi[i+ni  ] = (float) xi[i][1];
	h_xi[i+ni*2] = (float) xi[i][2];
	h_xi[i+ni*3] = (float) eps2;
      }
      CUDA_SAFE_CALL( hipMemcpy( d_xi, h_xi, ip_size, hipMemcpyHostToDevice) );


      dim3 grid((ni+NPIPE-1)/NPIPE); // ** Technic ** 
      dim3 threads  (NPIPE);
      {
	// type00:Id, N, sec, Gflop/s{max, avg, curr}: 202        131072  4.01055 651.291 651.113 651.118
	// type01:Id, N, sec, Gflop/s{max, avg, curr}: 202        131072  4.01369 650.777 650.627 650.608
	// basic :Id, N, sec, Gflop/s{max, avg, curr}: 202        131072  4.78496 545.754 545.738 545.739
	// nvidia:Id, N, sec, Gflop/s{max, avg, curr}: 202        131072  4.5376  575.526 575.504 575.489

	using namespace cunbody_kernel_type00;
	//	using namespace cunbody_kernel_basic;
	kernel<<< grid, threads >>>(d_xj, d_xi, d_fo, ni, nj1);

	//    using namespace cunbody_kernel_nvidia;
	//    kernel<<< grid, threads, (NJ_SHMEM*sizeof(float4)) >>>(d_xj, d_xi, d_fo, ni, nj1);

	CUT_CHECK_ERROR("Kernel execution failed");
      }

      // ------------------------------------------------------------ fo
      CUDA_SAFE_CALL( hipMemcpy( h_fo, d_fo, fo_size, hipMemcpyDeviceToHost) );

      for(int i=0;i<ni; i++){
	a[i][0] = (double)h_fo[i];
	a[i][1] = (double)h_fo[i+ni];
	a[i][2] = (double)h_fo[i+ni*2];
      }
    }


  }; // class cunbody1 __END__
}; // namespace libcunbody __END__  ----------------------------------------------------------------------------------

//-----------------------------------------------------------------------------------------------------------------
extern "C" void copyright_cunbody1(void);

void copyright_cunbody1(void)
{
  printf("Copyright(C) 2007 by Tsuyoshi Hamada <hamada@progrape.jp>, All rights reserved.\n");
}


static double T_Force = 0.0;
static double T_Write = 0.0;
static double T_Calc  = 0.0;
static double T_Read  = 0.0;
static unsigned ni_sum = 0;
static unsigned nj_sum = 0;
static double n_inter = 0.0;


extern "C" void cunbody_dumptime(void);

void
_cunbody_dumptime(void)
{
  printf("----------------------------------\n");
  printf("Time: %f |\t W(%f)\t C(%f)\t R(%f)\n",T_Force, T_Write, T_Calc, T_Read);
  printf("MB/s: %f \t %f\n",
	 (1.0e-6)*(sizeof(float)*ni_sum*IDIM+sizeof(float)*nj_sum*JDIM)/T_Write,
	 (1.0e-6)*(sizeof(float)*ni_sum*FDIM)/T_Read);

  printf("Gflop/s: %f \n",(1.0e-9)*n_inter*38.0/T_Calc);
  printf("Gflop/s: %f \n",(1.0e-9)*n_inter*38.0/T_Force);
  printf("n inter %e \n", n_inter);
  printf("\n");

  T_Force = T_Write = T_Calc = T_Read = 0.0;
  ni_sum = 0;
  nj_sum = 0;
  n_inter = 0.0;
}

void
cunbody_dumptime(void){ }

//-------------------------------------------------------------------

#define MAX_OMP_THRE (8)
static libcunbody::cunbody1_bench cunObj[MAX_OMP_THRE];


extern "C" void cunbody1_force_mp(int devid, double xj[][3], double mj[], double xi[][3], double eps2, double a[][3], int ni, int nj)
{
  using namespace std;
  using namespace libcunbody;
  cunObj[devid].set_devid(devid);
  cunObj[devid].force(xj, mj, xi, eps2, a, ni, nj);
}

extern "C" void cunbody1_force(double xj[][3], double mj[], double xi[][3], double eps2, double a[][3], int ni, int nj)
{
  using namespace std;
  using namespace libcunbody;
  int devid = 0;
  cunObj[devid].set_devid(devid);
  cunObj[devid].force(xj, mj, xi, eps2, a, ni, nj);
}



